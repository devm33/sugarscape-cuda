#include "hip/hip_runtime.h"
//Devraj Mehta
//Sugarscape
//Applies CUDA to ABM in Sugarscape

//Using pinned memory

//standard imports
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

//OpenGL imports
#include <GL/glew.h>
#include <GLUT/glut.h> //for Mac

//preprocessor definitions
#define R 50 //radius of mouse agent

//agent model
typedef struct {
	int x;
	int y;
	float sugar;
	float metabolism;
	int vision;
} Agent;

//variable declarations
int sugar_g, agent_g; //the number of blocks needed for sugar and agent kernel
int sugar_b, agent_b; //number of threads per block
float *sugar_levels, *sugar_maximums; //flattened matrices on host
float *sl_d, *sm_d; //matrices copied to device
float fps; //frames per second: epoque
char wtitle[256]; //title of glut window
Agent *agent_list, *a_d; //list of agents in world on host and device
int *agent_matrix, *am_d; //matrix of agent locations
int freeze_flag; //flag to halt all updating
long long int step; //counter for the number of iterations
int num_steps;
int W; //latteral resolution of world
int H; //vertical resolution of world
int N; //number of agents

//variables used in gl
double xmin,ymin,xmax,ymax;
int w, h; //screen size

//gaussian function to determine layout of sugar
float gauss(int x, int y, int x0, int y0, int sx, int sy)
{
	return expf(-0.5*(x-x0)*(x-x0)/sx/sx)*expf(-0.5*(y-y0)*(y-y0)/sy/sy);
}

//kernel to grow sugar patches at each time step
__global__ void grow_sugar(float *s_levels, float *s_maximums)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x; //index of sugar cell
	float growth_rate = 0.1f;
	s_levels[i] += growth_rate;
	if(s_levels[i] > s_maximums[i])
		s_levels[i] = s_maximums[i];
/*	if(s_levels[i] > 0.5 && s_maximums[i] < 0.7) //making a ridge
		s_levels[i] = 0.5;
	if(s_levels[i] > 0.7 && s_maximums[i] < 0.9)
		s_levels[i] = 0.7;*/
}

//kernel to updatae the agents' sugar levels
__global__ void feed_agents(Agent *a_list, int *a_mat, float *s_levels,
	int width, int height)
{
//declare variables
	int k; //agent's index
	float p; //amount of sugar agent can eat

//set index
	k = blockIdx.x * blockDim.x + threadIdx.x;

//if the agent is alive (this is bad for cuda simd)
	if(a_list[k].sugar > 0.0) {

//increment metabolism
		a_list[k].sugar -= a_list[k].metabolism;
//check if agent survived
		if(a_list[k].sugar <= 0.0) {
			a_mat[width * a_list[k].x + a_list[k].y] = -1;
		}
		else {
//if stil alive take sugar from current patch
			p = 1.0 - a_list[k].sugar;
			if(p > s_levels[width * a_list[k].x + a_list[k].y]) {
				a_list[k].sugar += s_levels[width * a_list[k].x + a_list[k].y];
				s_levels[width * a_list[k].x + a_list[k].y] = 0.0;
			}
			else {
				s_levels[width * a_list[k].x + a_list[k].y] -= p;
				a_list[k].sugar = 1.0;
			}
		}
	}
}

//kernel to update the agents' location
__global__ void move_agents(Agent *a_list, int *a_mat, float *s_levels,
		int width, int height)
{

//declare variables
	int i, j, k, x, y, f; //k: index of agent
	float b; //best sugar level seen
	int bx, by; //chosen location of best sugar level seen
	int v; //agent's vision
	k = blockIdx.x * blockDim.x + threadIdx.x;

//if the agent is alive (this is kinda bad for cuda simt)
	if(a_list[k].sugar > 0.0) {
		f=1;
		x = a_list[k].x;
		y = a_list[k].y;
		v = a_list[k].vision;
		while(f) {
			f=0;
			b = s_levels[width*x+y]; //best known sugar level
			bx = x; by = y;
			for(i = -v; i <= v /*&& b <= s*/; i++) {
				if(i+x >=0 && i+x < width) {
					for(j = -v; j <= v /*&& b <= s*/; j++) {
						if(j+y < height && j+y >= 0 && a_mat[width*(i+x)+j+y] == -1) { //check valid & vacant
							if(s_levels[width*(i+x)+j+y] > b) {
								b = s_levels[width*(i+x)+j+y];
								bx = i+x;
								by = j+y;
							}
						}
					}
				}
			}

			//move to location
			if(a_mat[width*bx+by] == k); //simply dont move
			else if(atomicExch(a_mat+width*bx+by, k) == -1) { //atomic test and set operation
				a_mat[width * a_list[k].x + a_list[k].y] = -1;
				a_list[k].x = bx;
				a_list[k].y = by;
			}
			else
				f=1;
		}
	}
}

//method to display world in opengl
void display(void)
{
	if(step==num_steps) {
//exit program and release memory
		hipHostFree(sugar_levels);
		free(sugar_maximums);
		hipHostFree(agent_list);
		hipHostFree(agent_matrix);
		hipFree(sl_d);
		hipFree(sm_d);
		hipFree(a_d);
		hipFree(am_d);
		exit(0);
		}

//check to see if permitted
	if(freeze_flag)
		return;

//declare variables
	int x,y,z;
	long long int n_left=0;
	double a,b,c,d;
	hipError_t cet;

//begin time
	fps = (float)clock()/CLOCKS_PER_SEC;

//run instructions on device
    grow_sugar<<< sugar_g, sugar_b>>>(sl_d, sm_d); //asynchronus, nonblocking

//block until all threads finish
	hipDeviceSynchronize();
	
//run instructions on device
    feed_agents<<< agent_g, agent_b>>>(a_d, am_d, sl_d, W, H); //asynchronus, nonblocking

//block until all threads finish
	hipDeviceSynchronize();
		
//run instructions on device
    move_agents<<< agent_g, agent_b>>>(a_d, am_d, sl_d, W, H); //asynchronus, nonblocking

//block until all threads finish
	hipDeviceSynchronize();

//copy updated matrices from device to host
	hipMemcpy(sugar_levels,	sl_d,	W*H*sizeof(float),	hipMemcpyDeviceToHost);
	hipMemcpy(agent_matrix,	am_d,	W*H*sizeof(int),	hipMemcpyDeviceToHost);
	//hipMemcpy(agent_list,		a_d,	N*sizeof(Agent),	hipMemcpyDeviceToHost);

//display world
	glClear(GL_COLOR_BUFFER_BIT);
	glBegin(GL_POINTS);
	for(x=0; x<w; x++) {
		for(y=0; y<h; y++) {
			z = W * (W * x / w) + H * y /h; //coordinates translated for sugarscape
			a = sugar_levels[z];
			d = 0.0;
			if(agent_matrix[z] != -1) {
				d = 1.0;
				n_left++;
			}
			glColor3f(0.0, a, d);
			b = xmax * x / w - xmin; //coordinates translated for gl
			c = ymax * y / h - ymin;
			glVertex2f(b,c);
		}
	}
	glEnd();
	
//scale counter
	if(W<w && H<h)
		n_left = (W*H)*n_left/(w*h);
	else if(W>w || H>h)
		n_left = N;

//finish and display epoque
	fps = 1.0 / ((float)clock()/CLOCKS_PER_SEC - fps);
	printf("%f\n", 1.0f/ fps);
	sprintf(wtitle, "Sugarscape (GPU) %d x %d  %lld agents  %3.1f fps  step #%lld",W,H,n_left,fps, step);
	glutSetWindowTitle(wtitle);

//end rendering and display updated buffer contents
	glutSwapBuffers();

//increment counter
	step++;

//check for errors in cuda
	cet = hipGetLastError();
	if(cet != hipSuccess)
		printf("CUDA ERROR: %s\n", hipGetErrorString(cet));
}

//sets all agents to initial states
void reset_agents()
{
	int j, k, m, n; //temp iter var
	for(k=0; k<W*H; k++)
		agent_matrix[k] = -1;
	for(k=0; k<N; k++) {
		j=1;
		while(j) {
			m = random() % H;
			n = random() % W;
			if(agent_matrix[ W * m + n] == -1)
				j=0;
		}
		agent_matrix[W * m + n] = k;
		agent_list[k].x = m;
		agent_list[k].y = n;
		agent_list[k].sugar = 1.0; 
		agent_list[k].metabolism = 0.001 * (random() % 900) + 0.1;
		agent_list[k].vision = random() % 9 + 1;
	}
	hipMemcpy(a_d, agent_list,			N*sizeof(Agent),	hipMemcpyHostToDevice);
	hipMemcpy(am_d, agent_matrix,		W*H*sizeof(int),	hipMemcpyHostToDevice);
}

//zeros all sugar levels
void reset_sugar()
{
	int k; //temp iter var
	for(k=0;k<W*H;k++)
		sugar_levels[k] = 0;
	hipMemcpy(sl_d, sugar_levels, W*H*sizeof(float),	hipMemcpyHostToDevice);
}

//method to register opengl key events
void keyfunc(unsigned char key,int xscr,int yscr)
{
	if(key=='q')
	{
//exit program and release memory
		hipHostFree(sugar_levels);
		free(sugar_maximums);
		hipHostFree(agent_list);
		hipHostFree(agent_matrix);
		hipFree(sl_d);
		hipFree(sm_d);
		hipFree(a_d);
		hipFree(am_d);
		printf("\nq pressed; program exiting.\n");
		exit(0);
	}
	else if(key=='r')
	{
//reset all sugar levels to zero
		reset_agents();
		reset_sugar();
	}
	else if(key=='s')
	{
//reset all sugar levels to zero
		reset_sugar();
	}
	else if(key=='a')
	{
//randomize and reset all agents
		reset_agents();
	}
	else if(key=='p')
	{
//(un)freeze all updating
		freeze_flag = freeze_flag ? 0 : 1;
	}
}

//method to register opengl mouse events
void mouse(int button,int state,int xscr,int yscr)
{
	int j, k; //temp iter vars
 	if(button==GLUT_LEFT_BUTTON)
	{
 		if(state==GLUT_DOWN)
		{
//set clicked upon sugar level to zero
			for(j=W*xscr/w-R; j<W*xscr/w+R; j++)
				if(j<W && j>=0)
					for(k=H*(h-yscr)/h-R; k<=H*(h-yscr)/h+R; k++)
						if(k<H && k>=0)
							sugar_levels[W * j + k] = 0;
			hipMemcpy(sl_d, sugar_levels, W*H*sizeof(float), hipMemcpyHostToDevice);
			//glutPostRedisplay(); // callback
		}
	}
	else if(button==GLUT_RIGHT_BUTTON)
	{
		if(state==GLUT_DOWN)
		{
//print this cell's properties
			printf("(%d, %d)\n", W*xscr/w, H*(h-yscr)/h);
			printf("\tpatch:\n\t\tsugar\t%f\n\t\tmax\t%f\n", sugar_levels[W * (W*xscr/w) + (H*(h-yscr)/h)],
				   sugar_maximums[W * (W*xscr/w) + (H*(h-yscr)/h)]);
			if(agent_matrix[W * (W*xscr/w) + (H*(h-yscr)/h)]==-1)
				printf("\tagent:\n\t\tnone\n");
			else {
				printf("\tagent:\n\t\tvision\t%d\n\t\tmetab\t%f\n\t\tsugar\t%f\n",
					   agent_list[agent_matrix[W * (W*xscr/w) + (H*(h-yscr)/h)]].vision,
					   agent_list[agent_matrix[W * (W*xscr/w) + (H*(h-yscr)/h)]].metabolism,
					   agent_list[agent_matrix[W * (W*xscr/w) + (H*(h-yscr)/h)]].sugar);
			}
			printf("\tmatrix:\n\t\tindex\t%d\n\t\tvalue\t%d\n", W * (W*xscr/w) + (H*(h-yscr)/h), agent_matrix[W * (W*xscr/w) + (H*(h-yscr)/h)]);
		}
	}
}

//method to register opengl mouse movement events
void move(int xscr, int yscr)
{
	int j, k; //temp iter vars
//set clicked upon sugar level to zero
	for(j=W*xscr/w-R; j<W*xscr/w+R; j++)
		if(j<W && j>=0)
			for(k=H*(h-yscr)/h-R; k<=H*(h-yscr)/h+R; k++)
				if(k<H && k>=0)
					sugar_levels[W * j + k] = 0;
	hipMemcpy(sl_d, sugar_levels, W*H*sizeof(float), hipMemcpyHostToDevice);
}

//method to handle the screen being resized
void reshape(int wscr,int hscr)
{
	w=wscr; h=hscr;
	glViewport(0,0,(GLsizei)w,(GLsizei)h);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	
	xmin=ymin=0.0; xmax=ymax=1.0;
	if(w<=h)
		ymax=1.0*(GLfloat)h/(GLfloat)w;
	else
		xmax=1.0*(GLfloat)w/(GLfloat)h;
	
	gluOrtho2D(xmin,xmax,ymin,ymax);
	glMatrixMode(GL_MODELVIEW);
}

//main method
int main(int argc, char* argv[])
{
//fetching for input
	if(argc!=3)
	{
		printf("please input N then number of steps\n");
		return 0;
	}
	N = atoi(argv[1]);
	W = N; H = N;
	w = N; h = N;
	N = N*N;
	num_steps = atoi(argv[2]);

//declare variables
	int i, j, k, l; //temp iter vars
	hipDeviceProp_t dp; //properties for device
	int max_threads; //the maximum number of threads per block

//set best device (the one with the most multiprocessors)
	hipGetDeviceCount(&i);
	k=0; l=0;
	for(j=0; j<i; j++) {
		hipGetDeviceProperties(&dp, j);
		if(dp.multiProcessorCount > l) {
			l = dp.multiProcessorCount;
			k = j;
		}
	}
	hipSetDevice(k);
	hipGetDeviceProperties(&dp, k);
	printf("Operating on %s\n", dp.name);

//define variables
	max_threads = dp.maxThreadsPerBlock;
//find the smallest x so that x*y=N, y<M, & x & y are both integers: perhaps there is a better way
	sugar_b = max_threads;
	while((W*H) % sugar_b != 0)
		sugar_b--;	
	sugar_g = W*H/sugar_b;
	agent_b = max_threads;
	while(N % agent_b != 0)
		agent_b--;
	agent_g = N/agent_b;
	freeze_flag = 0;
	step = 0;

//allocate matrices
	hipHostMalloc((void**)&sugar_levels, W*H*sizeof(float));
	hipMalloc((void**)&sl_d,		W*H*sizeof(float));

	sugar_maximums = (float*)malloc(	W*H*sizeof(float));
	hipMalloc((void**)&sm_d,		W*H*sizeof(float));

	hipHostMalloc((void**)&agent_list, N*sizeof(Agent));
	hipMalloc((void**)&a_d,		N*sizeof(Agent));

	hipHostMalloc((void**)&agent_matrix, W*H*sizeof(int));
	hipMalloc((void**)&am_d,		W*H*sizeof(int));

//initialize matrices on host
	memset(sugar_levels, 0, W*H*sizeof(float));
	for(i=0;i<W;i++)
		for(j=0;j<H;j++)
			sugar_maximums[W*i+j] = gauss(i,j,W/4,H*3/4,W/5,H/5) + gauss(i,j,W*3/4,H/4,W/5,H/5);
	reset_sugar();
	reset_agents();
	
//copy matrices to device
	hipMemcpy(sm_d, sugar_maximums,	W*H*sizeof(float), hipMemcpyHostToDevice);
	
//setup OpenGL
	glutInit(&argc,argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB);
	glutInitWindowSize(w,h);
	glutInitWindowPosition(0,0);
	glutCreateWindow("Sugarscape");
	glClearColor(1.0,1.0,1.0,0.0);
	
//gl callback functions
   	glutDisplayFunc(display);
	glutIdleFunc(display);
  	glutMouseFunc(mouse);
	glutMotionFunc(move);
 	glutKeyboardFunc(keyfunc);
	glutReshapeFunc(reshape);

//begin looping sugarscape
	glutMainLoop();
	
	return 0;
}
